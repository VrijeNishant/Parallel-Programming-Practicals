#include "hip/hip_runtime.h"

#include <Timer.hpp>
#include <iostream>
#include <iomanip>

using LOFAR::NSTimer;
using std::cout;
using std::cerr;
using std::endl;
using std::fixed;
using std::setprecision;

#define        OFFLOAD_KERNEL_HISTOGRAM                1

#define        THREADS_PER_BLOCK_X        16
#define        THREADS_PER_BLOCK_Y        16

 __global__ void 
kernel_optimized5( unsigned char *grayImage,long size, unsigned int *histogram, unsigned int height, unsigned int width )
{   
    __shared__ unsigned int temp[256];
    int index = (threadIdx.x * blockDim.y) + threadIdx.y;
    if (index < 256)
	temp[index] = 0;
     int x = (threadIdx.x + blockIdx.x * blockDim.x);
     int y = (threadIdx.y + blockIdx.y * blockDim.y);
    
     if (x >= width || y >= height) return; 		
     int i = y * width + x;	

  float grayPix = 0.0f;

  float r = static_cast< float >(grayImage[i]);
  float g = static_cast< float >(grayImage[(width * height) + i]);
  float b = static_cast< float >(grayImage[(2 * width * height) + i]);

  grayPix = __fadd_rn( __fadd_rn(__fadd_rn(__fmul_rn(0.3f, r),__fmul_rn(0.59f, g)), __fmul_rn(0.11f, b)), 0.5f);

  atomicAdd( &temp[static_cast< unsigned char >(grayPix)], 1);
  __syncthreads();

  if (index < 256 && temp[index] > 0){
	atomicAdd(&histogram[index],temp[index]);
  } 
}

void histogram1D(const int width, const int height, const unsigned char * inputImage, unsigned char * grayImage, unsigned int * histogram, unsigned char * histogramImage) {
	hipError_t devRetVal = hipSuccess; 
	NSTimer kernelTime = NSTimer("histogram", false, false);
	NSTimer memoryTime = NSTimer("memoryTime", false, false);
	NSTimer globalTime = NSTimer("globalTime", false, false);


	// start of computation
	globalTime.start();

	unsigned char *devInputImage;
	unsigned int *devHistogram;
	//unsigned int *devHistogram;

        int iImageSize = height * width * sizeof(unsigned char);

	         devRetVal = hipMalloc((void**)&devHistogram, sizeof(unsigned int) * 256);
        if (hipSuccess != devRetVal)
        {
                cout << "Cannot allocate memory" << endl;
                return;
        }

	 devRetVal = hipMalloc((void**)&devInputImage, 3*iImageSize);
        if (hipSuccess != devRetVal)
        {
                cout << "Cannot allocate memory" << endl;
                return;
        }


	memoryTime.start();
        devRetVal = hipMemcpy(devInputImage, inputImage, 3*iImageSize, hipMemcpyHostToDevice);

        if (hipSuccess != devRetVal)
        {

                cout << "Cannot copy memory";
                hipFree(devInputImage);
                return;
        }


	 if ( ( hipMemset(devHistogram, 0, 256 * sizeof(unsigned int))) != hipSuccess ) {
               cout << "Error in function memset." << endl;
               return ;
       }


       memoryTime.stop();

       dim3 dimBlock(THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y);

       int blockWidth = static_cast<unsigned int>(ceil(width / static_cast<float>(THREADS_PER_BLOCK_X)));

       int blockHeight = static_cast<unsigned int>(ceil(height / static_cast<float>(THREADS_PER_BLOCK_Y)));


       dim3 dimGrid(blockWidth, blockHeight);

	kernelTime.start();
//	kernel_darkGray<<<dimGrid, dimBlock>>>(width, height, devInputImage, devGrayImage);
//	kernel_histogram1D<<<dimGrid, dimBlock>>>(devInputImage, devGrayImage, devHistogram, width, height);
	kernel_optimized5<<<dimGrid, dimBlock>>>( devInputImage, iImageSize, devHistogram, height, width );  

    
	
      hipDeviceSynchronize();
        kernelTime.stop();

	 if ((devRetVal = hipGetLastError()) != hipSuccess)
        {
                cerr << "Uh, the kernel had some kind of issue: " << devRetVal << endl;
                hipFree(devInputImage);
                return;
        }

	memoryTime.start();


	devRetVal = hipMemcpy(histogram, devHistogram, 256 * sizeof(unsigned int),  hipMemcpyDeviceToHost);
        if (hipSuccess != devRetVal)
        {
                cout << "Cannot copy memory";
                hipFree(devHistogram);
           

                return;
        }
//	int i;
//	for (i = 0; i < 256; i ++)
//		cout << histogram[i] << endl;
        memoryTime.stop();

	hipFree(devInputImage);
	hipFree(devHistogram);

	globalTime.stop();
	//end of computation	


	 // Time GFLOP/s GB/s
        cout << fixed << setprecision(6) << kernelTime.getElapsed() << endl;
        cout << fixed << setprecision(6) << memoryTime.getElapsed() << endl;
        cout << fixed << setprecision(6) << globalTime.getElapsed() << endl;




}
